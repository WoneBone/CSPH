/*
 * Copyright 2020 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO LICENSEE:
 *
 * This source code and/or documentation ("Licensed Deliverables") are
 * subject to NVIDIA intellectual property rights under U.S. and
 * international Copyright laws.
 *
 * These Licensed Deliverables contained herein is PROPRIETARY and
 * CONFIDENTIAL to NVIDIA and is being provided under the terms and
 * conditions of a form of NVIDIA software license agreement by and
 * between NVIDIA and Licensee ("License Agreement") or electronically
 * accepted by Licensee.  Notwithstanding any terms or conditions to
 * the contrary in the License Agreement, reproduction or disclosure
 * of the Licensed Deliverables to any third party without the express
 * written consent of NVIDIA is prohibited.
 *
 * NOTWITHSTANDING ANY TERMS OR CONDITIONS TO THE CONTRARY IN THE
 * LICENSE AGREEMENT, NVIDIA MAKES NO REPRESENTATION ABOUT THE
 * SUITABILITY OF THESE LICENSED DELIVERABLES FOR ANY PURPOSE.  IT IS
 * PROVIDED "AS IS" WITHOUT EXPRESS OR IMPLIED WARRANTY OF ANY KIND.
 * NVIDIA DISCLAIMS ALL WARRANTIES WITH REGARD TO THESE LICENSED
 * DELIVERABLES, INCLUDING ALL IMPLIED WARRANTIES OF MERCHANTABILITY,
 * NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * NOTWITHSTANDING ANY TERMS OR CONDITIONS TO THE CONTRARY IN THE
 * LICENSE AGREEMENT, IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY
 * SPECIAL, INDIRECT, INCIDENTAL, OR CONSEQUENTIAL DAMAGES, OR ANY
 * DAMAGES WHATSOEVER RESULTING FROM LOSS OF USE, DATA OR PROFITS,
 * WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE OR OTHER TORTIOUS
 * ACTION, ARISING OUT OF OR IN CONNECTION WITH THE USE OR PERFORMANCE
 * OF THESE LICENSED DELIVERABLES.
 *
 * U.S. Government End Users.  These Licensed Deliverables are a
 * "commercial item" as that term is defined at 48 C.F.R. 2.101 (OCT
 * 1995), consisting of "commercial computer software" and "commercial
 * computer software documentation" as such terms are used in 48
 * C.F.R. 12.212 (SEPT 1995) and is provided to the U.S. Government
 * only as a commercial end item.  Consistent with 48 C.F.R.12.212 and
 * 48 C.F.R. 227.7202-1 through 227.7202-4 (JUNE 1995), all
 * U.S. Government End Users acquire the Licensed Deliverables with
 * only those rights set forth herein.
 *
 * Any use of the Licensed Deliverables in individual and commercial
 * software must include, in the user documentation and internal
 * comments to the code, the above Disclaimer and U.S. Government End
 * Users Notice.
 */

#include <hipblas.h>
#include <hip/hip_runtime.h>


#include <iostream>
#include <stdexcept>

#define DEBUG
#ifdef DEBUG
#define cudaCheckError(ans)                \
  {                                        \
    cudaAssert((ans), __FILE__, __LINE__); \
  }
inline void cudaAssert(hipError_t code, const char *file, int line, bool abort = true) {
  if (code != hipSuccess) {
    fprintf(stderr, "CUDA Error: %s at %s:%d\n",
            hipGetErrorString(code), file, line);
    if (abort)
      exit(code);
  }
}
// cublas API error checking
#define CUBLAS_CHECK(err)                                                  \
  do {                                                                     \
    hipblasStatus_t err_ = (err);                                           \
    if (err_ != HIPBLAS_STATUS_SUCCESS) {                                   \
      std::printf("cublas error %d at %s:%d\n", err_, __FILE__, __LINE__); \
      throw std::runtime_error("cublas error");                            \
    }                                                                      \
  } while (0)
#else
#define cudaCheckError(ans) ans
#define CUBLAS_CHECK(ans) ans
#endif

// define col major access
#define IDX2C(i, j, ld) (((j) * (ld)) + (i))

void printCudaInfo() {
  int deviceCount = 0;
  hipError_t err = hipGetDeviceCount(&deviceCount);

  printf("---------------------------------------------------------\n");
  printf("Found %d CUDA devices\n", deviceCount);

  for (int i = 0; i < deviceCount; i++) {
    hipDeviceProp_t deviceProps;
    hipGetDeviceProperties(&deviceProps, i);
    printf("Device %d: %s\n", i, deviceProps.name);
    printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
    printf("   Global mem: %.0f MB\n",
           static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
    printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
  }
  printf("---------------------------------------------------------\n");
}

/**
 * @brief Performs matrix multiplication using cuBLAS.
 *
 * This function multiplies two matrices A and B, and stores the result in matrix C.
 * The dimensions of the matrices are specified by m, n, and k.
 *
 * @param A Pointer to the first input matrix (m x k).
 * @param B Pointer to the second input matrix (k x n).
 * @param C Pointer to the output matrix (m x n).
 * @param m M dimension.
 * @param n N dimension.
 * @param k K dimension.
 * @param computeType The compute type to be used by cuBLAS (see slides for more information).
 * @param mode String to help identify the type of computation (use "FP32", "FP16" or "TF32" in the appropriate functions).
 * @param warm_up If true, performs a warm-up run before the actual computation where the timings are not considered.
 */

void cublas_gemm(float *A, float *B, float *C, int m, int n, int k, int lda, hipblasComputeType_t computeType, const char *mode, bool warm_up = false) {
  // TODO: TASK 10
  // Complete this base function
  int *devA, *devB, *devC;
  
  hipMalloc(&devA, (m*k) * sizeof(int));
  hipMalloc(&devB, (n*k) * sizeof(int));
  hipMalloc(&devC, (m*n) * sizeof(int));
  // Here you should copy host input matrices to the device

  hipMemcpy(devA, A,  (m*k) * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(devB, B,  (n*k) * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(devC, C,  (m*n) * sizeof(int), hipMemcpyHostToDevice);

  // Here you should create a handle for cuBLAS and initialize it with cublasCreate()
  hipblasHandle_t handle;
  hipblasCreate(&handle);
  // Here you should define whether the matrices are transposed or not
  hipblasOperation_t transa = HIPBLAS_OP_N;
  hipblasOperation_t transb = HIPBLAS_OP_N;

  // Scale factors are initialized: alpha = 1, beta = 0 for C = A * B
  // DO NOT MODIFY THESE VALUES
  const float alpha = 1.f;
  const float beta = 0.f;

  /* DO NOT MODIFY THIS PART
   * This part of the code is responsible for accurately measuring the time taken by the kernel.
   * The kernel is executed between the start and stop events.
   */
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

  // Here you should call the cuBLAS function that performs the matrix multiplication cublasGemmEx()
  // Do not forget to wrap it with the CUBLAS_CHECK() macro to detect errors
  CUBLAS_CHECK(hipblasGemmEx(handle, transa, transb, m, n, k, &alpha, devA, HIP_R_32F, m, devB, HIP_R_32F, k, &beta, devC,HIP_R_32F, m, computeType, HIPBLAS_GEMM_DEFAULT));

  /* DO NOT MODIFY THIS PART
   * This part of the code is responsible for accurately measuring the time taken by the kernel.
   * Here the time is recorded and printed.
   * The performance is calculated in GFLOPS.
   */
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  float elapsedTime;
  hipEventElapsedTime(&elapsedTime, start, stop);
  hipEventDestroy(start);
  hipEventDestroy(stop);
  if (!warm_up) {
    printf("Kernel Time (%s): %f ms\n", mode, elapsedTime);
    printf("GFLOPS (%s): %f\n", mode, 2.0 * m * n * k / (elapsedTime * 1e-3) / 1e9);
  }

  // Here you should copy the result matrix back to the host
  hipMemcpy(C, devC,  (m*n) * sizeof(int), hipMemcpyDeviceToHost);
  // Do not forget to free the memory on the device here
  hipFree(devA);
	hipFree(devB);
	hipFree(devC);
  // Do not forget to destroy the cublas handle here with the cublasDestroy() function
  hipblasDestroy(handle);
  return;
}

void cublas_gemm_fp32(float *A, float *B, float *C, int m, int n, int k, int lda, bool warm_up = false) {
  // Call the cublas gemm function for FP32 using cuda cores
  cublas_gemm(A, B, C, m, n, k, lda, HIPBLAS_COMPUTE_32F, "FP32", warm_up);
  return;
}

void cublas_gemm_fp16(float *A, float *B, float *C, int m, int n, int k, int lda, bool warm_up = false) {
  // TODO: TASK 10
  // Complete function so that GEMM executes with Tensor Cores using FP16
  cublas_gemm(A, B, C, m, n, k, lda,  HIPBLAS_COMPUTE_32F_FAST_16F, "FP16", warm_up);

  return;
}

void cublas_gemm_4xfp32(float *A, float *B, float *C, int m, int n, int k, int num_partitions, bool warm_up = false) {
  hipblasComputeType_t computeType = HIPBLAS_COMPUTE_32F;

  // Here you should allocate the memory on the device
  float *d_A, *d_B, *d_C;

  cudaCheckError(hipMalloc((void **)&d_A, m * k * sizeof(float)));
  cudaCheckError(hipMalloc((void **)&d_B, k * n * sizeof(float)));
  cudaCheckError(hipMalloc((void **)&d_C, m * n * sizeof(float)));

  // Here you should copy host input matrices to the device
  cudaCheckError(hipMemcpy(d_A, A, m * k * sizeof(float), hipMemcpyHostToDevice));
  // cublasSetMatrix(m, k, sizeof(float), A, lda, d_A, m);
  cudaCheckError(hipMemcpy(d_B, B, k * n * sizeof(float), hipMemcpyHostToDevice));

  // Here you should create a handle for cuBLAS and initialize it with cublasCreate()
  hipblasHandle_t handle;
  CUBLAS_CHECK(hipblasCreate(&handle));

  // Here you should define whether the matrices are transposed or not
  hipblasOperation_t transa = HIPBLAS_OP_N;
  hipblasOperation_t transb = HIPBLAS_OP_N;

  // Scale factors are initialized: alpha = 1, beta = 0 for C = A * B
  // DO NOT MODIFY THESE VALUES
  const float alpha = 1.f;
  const float beta = 0.f;

  /* DO NOT MODIFY THIS PART
   * This part of the code is responsible for accurately measuring the time taken by the kernel.
   * The kernel is executed between the start and stop events.
   */
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

  int partition_size = m / num_partitions;

  for (int i = 0; i < num_partitions; i++) {
    CUBLAS_CHECK(hipblasGemmEx(handle, transa, transb, partition_size, n,
                              k, &alpha, d_A + i * partition_size, HIP_R_32F, m, d_B,
                              HIP_R_32F, k, &beta, d_C + i * partition_size, HIP_R_32F,
                              m, computeType, HIPBLAS_GEMM_DEFAULT));
  }

  /* DO NOT MODIFY THIS PART
   * This part of the code is responsible for accurately measuring the time taken by the kernel.
   * Here the time is recorded and printed.
   * The performance is calculated in GFLOPS.
   */
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  float elapsedTime;
  hipEventElapsedTime(&elapsedTime, start, stop);
  hipEventDestroy(start);
  hipEventDestroy(stop);
  if (!warm_up) {
    printf("Kernel Time (4xFP32): %f ms\n", elapsedTime);
    printf("GFLOPS (4xFP32): %f\n", 2.0 * m * n * k / (elapsedTime * 1e-3) / 1e9);
  }

  // Here you should copy the result matrix back to the host
  cudaCheckError(hipMemcpy(C, d_C, m * n * sizeof(float), hipMemcpyDeviceToHost));

  // Do not forget to free the memory on the device here
  cudaCheckError(hipFree(d_A));
  cudaCheckError(hipFree(d_B));
  cudaCheckError(hipFree(d_C));

  // Do not forget to destroy the cublas handle here with the cublasDestroy() function
  CUBLAS_CHECK(hipblasDestroy(handle));

  return;
}

void cublas_gemm_mixed(float *A, float *B, float *C, int m, int n, int k, int num_partitions, bool warm_up = false) {
  // TODO: TASK 12
  // Correct function so that the error is zero and the performance is maximized

  hipblasComputeType_t computeType = HIPBLAS_COMPUTE_32F_FAST_16F;

  // Here you should allocate the memory on the device
  float *d_A, *d_B, *d_C;

  cudaCheckError(hipMalloc((void **)&d_A, m * k * sizeof(float)));
  cudaCheckError(hipMalloc((void **)&d_B, k * n * sizeof(float)));
  cudaCheckError(hipMalloc((void **)&d_C, m * n * sizeof(float)));

  // Here you should copy host input matrices to the device
  cudaCheckError(hipMemcpy(d_A, A, m * k * sizeof(float), hipMemcpyHostToDevice));
  // cublasSetMatrix(m, k, sizeof(float), A, lda, d_A, m);
  cudaCheckError(hipMemcpy(d_B, B, k * n * sizeof(float), hipMemcpyHostToDevice));

  // Here you should create a handle for cuBLAS and initialize it with cublasCreate()
  hipblasHandle_t handle;
  CUBLAS_CHECK(hipblasCreate(&handle));

  // Here you should define whether the matrices are transposed or not
  hipblasOperation_t transa = HIPBLAS_OP_N;
  hipblasOperation_t transb = HIPBLAS_OP_N;

  // Scale factors are initialized: alpha = 1, beta = 0 for C = A * B
  // DO NOT MODIFY THESE VALUES
  const float alpha = 1.f;
  const float beta = 0.f;

  /* DO NOT MODIFY THIS PART
   * This part of the code is responsible for accurately measuring the time taken by the kernel.
   * The kernel is executed between the start and stop events.
   */
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

  int partition_size = m / num_partitions;

  for (int i = 0; i < num_partitions; i++) {
    CUBLAS_CHECK(hipblasGemmEx(handle, transa, transb, partition_size, n,
                              k, &alpha, d_A + i * partition_size, HIP_R_32F, m, d_B,
                              HIP_R_32F, k, &beta, d_C + i * partition_size, HIP_R_32F,
                              m, computeType, HIPBLAS_GEMM_DEFAULT));
  }

  /* DO NOT MODIFY THIS PART
   * This part of the code is responsible for accurately measuring the time taken by the kernel.
   * Here the time is recorded and printed.
   * The performance is calculated in GFLOPS.
   */
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  float elapsedTime;
  hipEventElapsedTime(&elapsedTime, start, stop);
  hipEventDestroy(start);
  hipEventDestroy(stop);
  if (!warm_up) {
    printf("Kernel Time (Mixed): %f ms\n", elapsedTime);
    printf("GFLOPS (Mixed): %f\n", 2.0 * m * n * k / (elapsedTime * 1e-3) / 1e9);
  }

  // Here you should copy the result matrix back to the host
  cudaCheckError(hipMemcpy(C, d_C, m * n * sizeof(float), hipMemcpyDeviceToHost));

  // Do not forget to free the memory on the device here
  cudaCheckError(hipFree(d_A));
  cudaCheckError(hipFree(d_B));
  cudaCheckError(hipFree(d_C));

  // Do not forget to destroy the cublas handle here with the cublasDestroy() function
  CUBLAS_CHECK(hipblasDestroy(handle));

  return;
}
