#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <driver_functions.h>

#include <thrust/scan.h>
#include <thrust/device_ptr.h>
#include <thrust/device_malloc.h>
#include <thrust/device_free.h>

#include "CycleTimer.h"

#define DEBUG

#ifdef DEBUG
#define cudaCheckError(ans) { cudaAssert((ans), __FILE__, __LINE__); }
inline void cudaAssert(hipError_t code, const char *file, int line, bool abort=true){
  if (code != hipSuccess) {
    fprintf(stderr, "CUDA Error: %s at %s:%d\n", hipGetErrorString(code), file, line);
    if (abort) exit(code);
  }
}

#else
#define cudaCheckError(ans) ans
#endif

static inline int updiv(int n, int d) {
    return (n+d-1)/d;
}

// helper function to round an integer up to the next power of 2
static inline int nextPow2(int n) {
    n--;
    n |= n >> 1;
    n |= n >> 2;
    n |= n >> 4;
    n |= n >> 8;
    n |= n >> 16;
    n++;
    return n;
}

void exclusive_scan(int* input, int N, int* result, int threadsPerBlock);
double cudaScanThrust(int* inarray, int* end, int* resultarray);

///////////////////////////////////////
//// WRITE YOUR CUDA KERNELS HERE /////
///////////////////////////////////////

__global__ void  gpu_initX(int N, int* devX)
{
    // TODO
}

__global__ void gpu_makeZ(int N, int* devX, int* devY, int* devZ)
{
    // TODO
}

__device__ int gpu_condition(int i, int *A)
{
    // TODO
    return 0;
}

__global__ void gpu_makeW(int N, int* devZ, int* devW)
{
    // TODO
}

__global__ void gpu_find_pattern(int N, int* devW, int* devZ, int* output) 
{
    // TODO
}


////////////////////////////////////////////////////////////////////////////////
// runGPU 
/////////////////////////////////////////
// Timing wrapper around your complete GPU code. You should not modify this function.
int runGPU(int threadsPerBlock, int N, int* hostY, int *gpuX, int *gpuY, int *gpuZ, int *gpuW, int *gpuResult)
{
    // here we declare the arrays that we will need 
    int *devX, *devY;
    int *devZ, *devW;
    int *devResult;
    int gpuCount = 0;

    int numBlocks = updiv(N,threadsPerBlock);

    ////////////////////////////
    /// GPU DATA ALLOCATION ////
    ////////////////////////////
    /// TODO :: you should malloc devX, devY, devZ, devW, and devResult
    // all arrays are integer arrays of size of N 


    //////////////////////
    /// H2D TRANSFERS ////
    //////////////////////
    /// TODO: complete H2D for hostY to devY

    //////////////////////////////////
    /// YOUR CUDA KERNEL LAUNCHES ////
    //////////////////////////////////
    gpu_initX<<<numBlocks, threadsPerBlock>>>(N, devX);
    gpu_makeZ<<<numBlocks, threadsPerBlock>>>(N, devX, devY, devZ);
    gpu_makeW<<<numBlocks, threadsPerBlock>>>(N, devZ, devW);
    exclusive_scan(devW, N, devW, threadsPerBlock); //if you want to use it
    gpu_find_pattern<<<numBlocks, threadsPerBlock>>>(N, devW, devZ, devResult);
   
    //////////////////////
    /// H2D TRANSFERS ////
    //////////////////////
    // TODO :: You should copy back devX, devY, devZ, devW, devResults
    // into arrays gpuX, gpuX, gpuZ, gpuW, gpuResults, respectively 
 
    //////////////////
    /// CUDA FREE ////
    //////////////////
    // TODO:: You should free all arrays that you allocated before!

    return gpuCount;
}

////////////////////////////////////////////////////////////////
//// EXCLUSIVE SCAN AND HELP FUNCTIONS FROM LAB3 TUTORIALS /////
////////////////////////////////////////////////////////////////

void printCudaInfo()
{
    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++)
    {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n"); 
}



// exclusive_scan --
//
__global__ void upsweep_kernel(int N, int* output, int two_d, int two_dplus1) 
{
    size_t i = blockIdx.x*blockDim.x + threadIdx.x;
    size_t idx = two_dplus1*i;

    if(idx<N)
    {
        output[idx+two_dplus1-1]+=output[idx+two_d-1];
    }
}

__global__ void downsweep_kernel(int N, int* output, int two_d, int two_dplus1)
{
    size_t i = blockIdx.x*blockDim.x + threadIdx.x;
    size_t idx = two_dplus1*i;

    if(idx<N)
    {
        int t=output[idx+two_d-1];
        output[idx+two_d-1]=output[idx+two_dplus1-1];
        output[idx+two_dplus1-1]+=t;
    }
}


void exclusive_scan(int* input, int N, int* result, int threadsPerBlock)
{
    int numThreadBlocks;

    for(int two_d=1; two_d<nextPow2(N)/2; two_d*=2)
    {
        int two_dplus1=2*two_d;
        numThreadBlocks = updiv(nextPow2(N)/two_dplus1, threadsPerBlock); 
        upsweep_kernel<<<numThreadBlocks, threadsPerBlock>>>(nextPow2(N), result, two_d, two_dplus1);
        cudaCheckError(hipDeviceSynchronize());
    }

    cudaCheckError(hipMemset(result+nextPow2(N)-1, 0, sizeof(int)));

    for(int two_d=nextPow2(N)/2; two_d>=1; two_d/=2)
    {
        int two_dplus1=2*two_d;
        numThreadBlocks = updiv(nextPow2(N)/two_dplus1, threadsPerBlock);        
        downsweep_kernel<<<numThreadBlocks, threadsPerBlock>>>(nextPow2(N), result, two_d, two_dplus1);
        cudaCheckError(hipDeviceSynchronize());
    }
}

// cudaScanThrust --
//
double cudaScanThrust(int* inarray, int* end, int* resultarray) {

    int length = end - inarray;
    thrust::device_ptr<int> d_input = thrust::device_malloc<int>(length);
    thrust::device_ptr<int> d_output = thrust::device_malloc<int>(length);
    
    hipMemcpy(d_input.get(), inarray, length * sizeof(int), hipMemcpyHostToDevice);

    double startTime = CycleTimer::currentSeconds();

    thrust::exclusive_scan(d_input, d_input + length, d_output);

    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();
   
    hipMemcpy(resultarray, d_output.get(), length * sizeof(int), hipMemcpyDeviceToHost);

    thrust::device_free(d_input);
    thrust::device_free(d_output);

    double overallDuration = endTime - startTime;
    return overallDuration; 
}